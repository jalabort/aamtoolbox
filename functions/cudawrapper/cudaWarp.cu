
#include <hip/hip_runtime.h>
__global__ void cudaWarp(double * warpedImg, const double * indBase, const double * img, const double *resImg1, const double * tri, const double * nTri, const double * pixelTri, const double * x, const double * y, const double * alphas, const double * betas, const double * gammas, const double * idMax) 
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (id < (int) idMax[0]) 
    {
        int idTri = (int) pixelTri[id];

        int idTriVer1 = (int) tri[idTri] - 1;
        int idTriVer2 = (int) tri[idTri + (int) nTri[0]] - 1;
        int idTriVer3 = (int) tri[idTri + 2 * (int) nTri[0]] - 1;

        float x1 = x[idTriVer1] - 1;
        float x2 = x[idTriVer2] - 1;
        float x3 = x[idTriVer3] - 1;

        float y1 = y[idTriVer1] - 1;
        float y2 = y[idTriVer2] - 1;
        float y3 = y[idTriVer3] - 1;

        float alpha = alphas[id];
        float beta = betas[id];
        float gamma = gammas[id];

        float tX = x1 * gamma + x2 * alpha + x3 * beta;
        float tY = y1 * gamma + y2 * alpha + y3 * beta;

        int fTX = floor(tX);
        int fTY = floor(tY);
        int fTX1 = fTX + 1;
        int fTY1 = fTY + 1;

        float dX = tX - fTX;
        float dY = tY - fTY;
        float dX1 = 1 - dX;
        float dY1 = 1 - dY;

        float w1 = dX1 * dY1;
        float w2 = dX  * dY1;
        float w3 = dX1 * dY  ;
        float w4 = dX  * dY  ;

        int aux1 = fTX * resImg1[0];
        int aux2 = fTX1 * resImg1[0];

        int idImg1 = fTY  + aux1;
        int idImg2 = fTY  + aux2;
        int idImg3 = fTY1 + aux1;
        int idImg4 = fTY1 + aux2;

        int uv = (int) indBase[id];

        warpedImg[uv] =  w1 * img[idImg1] + w2 * img[idImg2] + w3 * img[idImg3] + w4 * img[idImg4];

    }

}